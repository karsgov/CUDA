#include <iostream>
#include <string>

#include <opencv2/opencv.hpp>
#include <hipDNN.h>

// Macro for checking status of cudnn objects
#define checkCUDNN(expression) {                               \
    hipdnnStatus_t status = (expression);                       \
    if (status != HIPDNN_STATUS_SUCCESS) {                      \
        std::cerr << "Error on line " << __LINE__ << ": "      \
                  << hipdnnGetErrorString(status) << std::endl; \
        std::exit(EXIT_FAILURE);                               \
    }                                                          \
}                                                              \

cv::Mat loadImage(const std::string imagePath) {
    cv::Mat image = cv::imread(imagePath, CV_LOAD_IMAGE_COLOR);
    image.convertTo(image, CV_32FC3);
    cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
    return image;
}

void saveImage(const std::string fname, float* buffer, int height, int width) {
    cv::Mat outputImage(height, width, CV_32FC3, buffer);
    // Clip negative values to zero
    cv::threshold(outputImage, 
                  outputImage,
                  /*threshold=*/0,
                  /*maxval=*/0,
                  cv::THRESH_TOZERO);
    cv::normalize(outputImage, outputImage, 0.0, 255.0, cv::NORM_MINMAX);
    outputImage.convertTo(outputImage, CV_8UC3);
    cv::imwrite(fname, outputImage);
    std::cout << "Saved image to " << fname << std::endl;
}
    
int main(int argc, char const *argv[]) {

    // Parse args
    if ( argc < 2) {
        std::cerr << "Usage:" << std::endl;
        std::cerr << "\t" << argv[0] << " <image-path> [gpu=0]" << std::endl;
        return -1;
    }

    int gpuId = (argc > 2) ? std::atoi(argv[2]) : 0;

    // Load Image
    cv::Mat image = loadImage(argv[1]);

    // Create CUDNN context
    hipdnnHandle_t cudnn;
    checkCUDNN(hipdnnCreate(&cudnn));
    
    // Describe input tensor
    hipdnnTensorDescriptor_t inputDescriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&inputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(inputDescriptor,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/1,
                                          /*channels=*/3,
                                          /*image_height=*/image.rows,
                                          /*image_width=*/image.cols));

    // Describe kernel tensor
    hipdnnFilterDescriptor_t kernelDescriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&kernelDescriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(kernelDescriptor,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*format=*/HIPDNN_TENSOR_NCHW,
                                          /*out_channels=*/3,
                                          /*in_channels=*/3,
                                          /*kernel_height=*/3,
                                          /*kernel_width=*/3));

    // Describe convolution kernel
    hipdnnConvolutionDescriptor_t convolutionDescriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolutionDescriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolutionDescriptor,
                                               /*pad_height=*/1,
                                               /*pad_width=*/1,
                                               /*vertical_stride=*/1,
                                               /*horizontal_stride=*/1,
                                               /*dilation_height=*/1,
                                               /*dilation_width=*/1,
                                               /*mode=*/HIPDNN_CROSS_CORRELATION));
                                               ///*computeType=*/HIPDNN_DATA_FLOAT));

    // Get output image dimensions
    int batchSize{0}, channels{0}, height{0}, width{0};
    checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolutionDescriptor,
                                                     inputDescriptor,
                                                     kernelDescriptor,
                                                     &batchSize,
                                                     &channels,
                                                     &height,
                                                     &width));

    // Describe output tensor
    hipdnnTensorDescriptor_t outputDescriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&outputDescriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(outputDescriptor,
                                          /*format=*/HIPDNN_TENSOR_NHWC,
                                          /*dataType=*/HIPDNN_DATA_FLOAT,
                                          /*batch_size=*/1,
                                          /*channels=*/3,
                                          /*image_height=*/image.rows,
                                          /*image_width=*/image.cols));
    
    // Describe convolution operation
    hipdnnConvolutionFwdAlgo_t convolutionAlgorithm;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                   inputDescriptor,
                                                   kernelDescriptor,
                                                   convolutionDescriptor,
                                                   outputDescriptor,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                   /*memoryLimitInBytes=*/0,
                                                   &convolutionAlgorithm));

    // Get workspace size
    size_t workspaceBytes = 0;
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       inputDescriptor,
                                                       kernelDescriptor,
                                                       convolutionDescriptor,
                                                       outputDescriptor,
                                                       convolutionAlgorithm,
                                                       &workspaceBytes));

    std::cout << "Input image : " <<  image.channels() << " x " << image.rows 
              << " x " << image.cols << std::endl;
    std::cout << "Output image : " << channels << " x " << height << " x " 
              << width << std::endl;
    std::cout << "Workspace Size : " << (workspaceBytes/1048576.0) << " MB" << std::endl;


    // Allocate memory
    void* d_workspace{nullptr};
    hipMalloc(&d_workspace, workspaceBytes);

    int imageBytes = batchSize * channels * height * width * sizeof(float);

    float* d_input{nullptr};
    hipMalloc(&d_input, imageBytes);
    hipMemcpy(d_input, image.ptr<float>(0), imageBytes, hipMemcpyHostToDevice);

    float* d_output{nullptr};
    hipMalloc(&d_output, imageBytes);
    hipMemset(d_output, 0, imageBytes);

    // Define 2d kernel 
    const float kernelTemplate[3][3] = {
        {1, 1, 1},
        {1, -8, 1},
        {1, 1, 1}  
    };
    
    // Assign same kernel to differnt channels
    float h_kernel[3][3][3][3];

    for(int n=0; n < 3; ++n) {
        for(int c=0; c < 3; ++c) {
            for(int h=0; h < 3; ++h) {
                for(int w=0; w < 3; ++w) {
                    h_kernel[n][c][h][w] = kernelTemplate[h][w];
                }
            }
        }
    }

    float *d_kernel{nullptr};
    hipMalloc(&d_kernel, sizeof(h_kernel));
    hipMemcpy(d_kernel, h_kernel, sizeof(h_kernel), hipMemcpyHostToDevice);


    // Finally run convolution operation
    const float alpha=1.0f, beta = 0.0f;

    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       inputDescriptor,
                                       d_input,
                                       kernelDescriptor,
                                       d_kernel,
                                       convolutionDescriptor,
                                       convolutionAlgorithm,
                                       d_workspace,
                                       workspaceBytes,
                                       &beta,
                                       outputDescriptor,
                                       d_output));

    // Copy output data from GPU to CPU memory
    float *h_output = new float[imageBytes];
    hipMemcpy(h_output, d_output, imageBytes, hipMemcpyDeviceToHost);

    // Save image
    saveImage("cuddn_out.png", h_output, height, width);

    // Free memory
    delete [] h_output;
    hipFree(d_kernel);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_workspace);

    // Destroy desciptors
    hipdnnDestroyTensorDescriptor(inputDescriptor);
    hipdnnDestroyTensorDescriptor(outputDescriptor);
    hipdnnDestroyFilterDescriptor(kernelDescriptor);
    hipdnnDestroyConvolutionDescriptor(convolutionDescriptor);

    // Destroy cudnn context
    hipdnnDestroy(cudnn);

    return 0;
}
